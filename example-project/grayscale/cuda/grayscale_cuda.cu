
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
typedef struct
{
     unsigned char red, green, blue;
} PPMPixel;
typedef struct
{
     int x, y;
     PPMPixel *data;
} PPMImage;
#define CREATOR "ParallelProgrammer"
#define RGB_COMPONENT_COLOR 255
static PPMImage *ReadPPM(const char *filename)
{
     char buff[16];
     PPMImage *img;
     FILE *fp;
     int c, rgb_comp_color;
     fp = fopen(filename, "rb");
     if (!fp)
     {
          fprintf(stderr, "Unable to open file '%s'\n", filename);
          exit(1);
     }
     if (!fgets(buff, sizeof(buff), fp))
     {
          perror(filename);
          exit(1);
     }
     if (buff[0] != 'P' || buff[1] != '6')
     {
          fprintf(stderr, "Invalid image format (must be 'P6')\n");
          exit(1);
     }
     img = (PPMImage *)malloc(sizeof(PPMImage));
     if (!img)
     {
          fprintf(stderr, "Unable to allocate memory\n");
          exit(1);
     }
     c = getc(fp);
     while (c == '#')
     {
          while (getc(fp) != '\n')
               ;
          c = getc(fp);
     }
     ungetc(c, fp);
     if (fscanf(fp, "%d %d", &img->x, &img->y) != 2)
     {
          fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
          exit(1);
     }
     if (fscanf(fp, "%d", &rgb_comp_color) != 1)
     {
          fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
          exit(1);
     }
     if (rgb_comp_color != RGB_COMPONENT_COLOR)
     {
          fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
          exit(1);
     }
     while (fgetc(fp) != '\n')
          ;
     img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

     if (!img)
     {
          fprintf(stderr, "Unable to allocate memory\n");
          exit(1);
     }
     if (fread(img->data, 3 * img->x, img->y, fp) != img->y)
     {
          fprintf(stderr, "Error loading image '%s'\n", filename);
          exit(1);
     }
     fclose(fp);
     return img;
}
void WritePPM(const char *filename, PPMImage *img)
{
     FILE *fp;
     fp = fopen(filename, "wb");
     if (!fp)
     {
          fprintf(stderr, "Unable to open file '%s'\n", filename);
          exit(1);
     }
     fprintf(fp, "P6\n");
     fprintf(fp, "# Created by %s\n", CREATOR);
     fprintf(fp, "%d %d\n", img->x, img->y);
     fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);
     fwrite(img->data, 3 * img->x, img->y, fp);
     fclose(fp);
}
__global__ void grayscale(int n, PPMPixel *source, PPMPixel *target)
{
     int id = blockIdx.x * blockDim.x + threadIdx.x;
     if (id < n)
     {
          double f = 1;
          double l = 0.3 * source[id].red + 0.6 * source[id].green + 0.1 * source[id].blue;
          target[id].red = source[id].red + f * (l - source[id].red);
          target[id].green = source[id].green + f * (l - source[id].green);
          target[id].blue = source[id].blue + f * (l - source[id].blue);
     }
}
void ChangeColorPPM(PPMImage *img)
{
     struct timeval tval_before, tval_after, tval_result;
     int N = img->x * img->y;
     PPMPixel *source, *d_source, *target, *d_target;
     source = (PPMPixel *)malloc(N * sizeof(PPMPixel));
     target = (PPMPixel *)malloc(N * sizeof(PPMPixel));
     hipMalloc(&d_source, N * sizeof(PPMPixel));
     hipMalloc(&d_target, N * sizeof(PPMPixel));
     gettimeofday(&tval_before, NULL);
     hipMemcpy(d_source, img->data, N * sizeof(PPMPixel), hipMemcpyHostToDevice);
     hipMemcpy(d_target, img->data, N * sizeof(PPMPixel), hipMemcpyHostToDevice);
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за копиране на масивите в устройството\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     gettimeofday(&tval_before, NULL);
     grayscale<<<(N + 383) / 384, 384>>>(N, d_source, d_target);
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за изпълнението на kernel функцията\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     gettimeofday(&tval_before, NULL);
     hipMemcpy(target, d_target, N * sizeof(PPMPixel), hipMemcpyDeviceToHost);
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за копиране на данните обратно в хоста\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     gettimeofday(&tval_before, NULL);
     for (int i = 0; i < N; i++)
     {
          img->data[i] = target[i];
     }
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за копиране на данните обратно в масива с пикселите\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     hipFree(d_source);
     hipFree(d_target);
     free(source);
     free(target);
}
int main(void)
{
     PPMImage *image;
     struct timeval tval_before, tval_after, tval_result;
     gettimeofday(&tval_before, NULL);
     image = ReadPPM("image.ppm");
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("%ld.%06ld     секунди за четене на данните от изображението\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     gettimeofday(&tval_before, NULL);
     ChangeColorPPM(image);
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  --------\n");
     printf("  %ld.%06ld   секунди за обработка на данните от изображението\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     gettimeofday(&tval_before, NULL);
     WritePPM("grayscale_cuda_result.ppm", image);
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("%ld.%06ld     секунди за запис на данните в изображението\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
}
